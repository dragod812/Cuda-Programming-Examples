#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime_api.h>
#include <list> 
#include <sstream>
#include <fstream>
#include <string> 
#include <stdio.h>
#include <iostream> 

#define FULL_MASK 0xffffffff

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

class Points
{
  float *X;
  float *Y;

public:
  __host__ __device__ Points() : X(NULL), Y(NULL) {}

  __host__ __device__ Points( float *x, float *y ) : X(x), Y(y) {}

  __host__ __device__ __forceinline__ float2 getPoint( int idx ) const
  {
    return make_float2( X[idx], Y[idx] );
  }

  __host__ __device__ __forceinline__ void setPoint( int idx, const float2 &p ) 
  {
    X[idx] = p.x;
    Y[idx] = p.y;
  }

  __host__ __device__ __forceinline__ void set( float *x, float *y )
  {
    X = x;
    Y = y;
  }
};


class Bounding_Box{
    float xMin , xMax, yMin, yMax;
public:
	__host__ __device__ Bounding_Box(){
		xMin = -700;
		yMin = -700;
		xMax = 700;
		yMax = 700;
	}
	__host__ __device__ void computeCenter( float2 &center ){
		center.x = 0.5f * ( m_p_min.x + m_p_max.x );
		center.y = 0.5f * ( m_p_min.y + m_p_max.y );
	}
	__host__ __device__ __forceinline__ float getxMax() const {
		return xMax;		
	}
	__host__ __device__ __forceinline__ float getyMax() const {
		return yMax;		
	}
	__host__ __device__ __forceinline__ float getyMin() const {
		return yMin;		
	}
	__host__ __device__ __forceinline__ float getxMin() const {
		return xMin;		
	}
	__host__ __device__ bool contains(const float2 &p) const {
		return (p.x >= xMin && p.y >= yMin && p.x < xMax && p.y < yMax);
	}
	__host__ __device__ void set(float x, float y, float X, float Y){
		xMin = x;
		yMin = y;
		xMax = X;
		yMax = y;
	}
};

class Quadtree_Node{
	//node index;
	int idx;
	Bounding_Box bb;
	//startIdx of points in the bb for the global data array
	int startIdx, endIdx;	
	Quadtree_Node *NE, *NW, *SW, *SE; 
public:
	__host__ __device__ Quadtree_Node() : idx(-1), startIdx(-1), endIdx(-1), NE(NULL), NW(NULL), SW(NULL), SE(NULL){

	}
	__host__ __device__ bool isNull(){
		return (idx == -1);
	}
	__host__ __device__ void setIdx(int idx){
		this->idx = idx;		
	}
	__host__ __device__ int getIdx(){
		return idx;
	}
	__host__ __device__ void setBoundingBox(float x,float y,float X,float Y){
		bb.set(x, y, X, Y);	
	}
	__host__ __device__ __forceinline__ Bounding_Box& getBoundingBox(){
		return bb;
	}
	__host__ __device__ void setRange(int s, int e){
		startIdx = s;
		endIdx = e;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getSW(){
		return SW;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getSE(){
		return SE;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getNW(){
		return NW; 
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getNE(){
		return NE; 
	}
	__host__ __device__ __forceinline__ void setSW( Quadtree_Node* ptr){
		SW = ptr;
	}
	__host__ __device__ __forceinline__ void setNW( Quadtree_Node* ptr){
		NW = ptr;
	}
	__host__ __device__ __forceinline__ void setSE( Quadtree_Node* ptr){
		SE = ptr;
	}
	__host__ __device__ __forceinline__ void setNE( Quadtree_Node* ptr){
		NE = ptr;
	}

	__host__ __device__ __forceinline__ int getStartIdx(){
		return startIdx;
	}
	__host__ __device__ __forceinline__ int getEndIdx(){
		return endIdx;
	}
 	__host__ __device__ __forceinline__ int numberOfPoints(){
		return endIdx - startIdx + 1;
	}
};

struct Random_generator
{
  __host__ __device__ unsigned int hash(unsigned int a)
  {
      a = (a+0x7ed55d16) + (a<<12);
      a = (a^0xc761c23c) ^ (a>>19);
      a = (a+0x165667b1) + (a<<5);
      a = (a+0xd3a2646c) ^ (a<<9);
      a = (a+0xfd7046c5) + (a<<3);
      a = (a^0xb55a4f09) ^ (a>>16);
      return a;
  }

  __host__ __device__ __forceinline__ thrust::tuple<float, float> operator()() 
  {
    unsigned seed = hash( blockidx.x*blockdim.x + threadidx.x );
    thrust::default_random_engine rng(seed);
    thrust::random::uniform_real_distribution<float> distrib;
    return thrust::make_tuple( distrib(rng), distrib(rng) );
  }
};

class Parameters
{
	const int min_points_per_node;
	//Introduced to minimise shifting of points
	//can have values only 0 and 1 based on slot
	//points[points_slot] is input slot
	//points[(points_slot+1)%2] is output slot
	int points_slot;
	__host__ __device__ Parameters( int mppn ) : min_points_per_node(mppn), points_slot(0) {}
	//copy constructor for the evaluation of children of current node
	__host__ __device__ Parameters( Parameters prm ) : min_points_per_node(prm.min_points_per_node), points_slot((prm.points_slot+1)%2) {}

}

template< int NUM_THREADS_PER_BLOCK >
__global__ 
void buildQuadtree( Quadtree_Node *root, Points *points, Parameters prmtrs){
	const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warpSize;

	//shared memory
	extern __shared__ int smem[];
	
	//warp_id and lane_id
	const int warp_id = threadIdx.x / warpSize;
	const int lane_id = threadIdx.x % warpSize;
	
	// Addresses of shared Memory
	volatile int *s_num_pts[4];
	for( int i = 0 ; i < 4 ; ++i )
		s_num_pts[i] = (volatile int *) &smem[i*NUM_WARPS_PER_BLOCK];

	int lane_mask_lt = (1 << lane_id) - 1; 
	
	int NUM_POINTS = root->numberOfPoints();

	//stop recursion if num_points <= minimum number of points required for recursion 
	if( NUM_POINTS <= prmtrs.min_points_per_node){

		//unable to understand the use of point_selector
		return;
	}

	//get Center of the bounding box
	float2 center;
	const Bounding_Box &box = root->getBoundingBox();
	box.computeCenter( center );

	int NUM_POINTS_PER_WARP = max( warpSize, ( NUM_POINTS + NUM_WARPS_PER_BLOCK - 1 ) / NUM_WARPS_PER_BLOCK );
	
	int warp_begin = root->getStartIdx() + warp_id*NUM_POINTS_PER_WARP;
	int warp_end = min(warp_begin + NUM_POINTS_PER_WARP, root->getEndIdx());

	if( lane_id == 0 )
	{
		s_num_pts[0][warp_id] = 0;
		s_num_pts[1][warp_id] = 0;
		s_num_pts[2][warp_id] = 0;
		s_num_pts[3][warp_id] = 0;
	}
	
	//input points
	const Points &input = points[prmtrs.points_slot];
	
	//__any_sync(unsigned mask, predicate):
		//Evaluate predicate for all non-exited threads in mask and return non-zero if and only if predicate evaluates to non-zero for any of them.
	//count points in each warp that belong to which child
	for( int itr = warp_begin + lane_id ; __any_sync(FULL_MASK, itr < warp_end ) ; itr += warpSize){
		bool is_active = itr < warp_end;

		//get the coordinates of the point
		float2 curP;
		if(is_active)
			curP = input.getPoint(itr);
		else
			curP = make_float2(0.0f, 0.0f);

		//consider standard anticlockwise quadrants for numbering 0 to 3

		//__ballot_sync(unsigned mask, predicate):
			//Evaluate predicate for all non-exited threads in mask and return an integer whose Nth bit is set if and only if predicate evaluates to non-zero for the Nth thread of the warp and the Nth thread is active.
		//__popc
			//Count the number of bits that are set to 1 in a 32 bit integer.
		//top-right Quadrant (Quadrant - I)
		int cnt = __popc( __ballot_sync(FULL_MASK, is_active && curP.x >= center.x && curP.y >= center.y));
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[0][warp_id] += cnt;

		//top-left Quadrant (Quadrant - II)
		cnt = __popc( __ballot_sync(FULL_MASK, is_active && curP.x < center.x && curP.y >= center.y));
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[1][warp_id] += cnt;

		//bottom-left Quadrant (Quadrant - III)
		cnt = __popc( __ballot_sync(FULL_MASK, is_active && curP.x < center.x && curP.y < center.y));
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[2][warp_id] += cnt;

		//bottom-right Quadrant (Quadrant - IV)
		cnt = __popc( __ballot_sync(FULL_MASK, is_active && curP.x >= center.x && curP.y < center.y));
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[3][warp_id] += cnt;
	}		

	//sychronize warps
	//__syncthreads() acts as a barrier at which all threads in the block must wait before any is allowed to proceed
	__syncthreads();

	// Scan the warps' results to know the "global" numbers.
	// First 4 warps scan the numbers of points per child (inclusive scan).
	if( warp_id < 4 )
	{
		int num_pts = lane_id < NUM_WARPS_PER_BLOCK ? s_num_pts[warp_id][lane_id] : 0;
		#pragma unroll
		for( int offset = 1 ; offset < NUM_WARPS_PER_BLOCK ; offset *= 2 )
		{
			int n = __shfl_up_sync( num_pts, offset, NUM_WARPS_PER_BLOCK );
			if( lane_id >= offset )
				num_pts += n;
		}
		if( lane_id < NUM_WARPS_PER_BLOCK )
			s_num_pts[warp_id][lane_id] = num_pts;
	}
	__syncthreads();
	// Compute global offsets.
	if( warp_id == 0 )
	{
		int sum = s_num_pts[0][NUM_WARPS_PER_BLOCK-1];
		for( int row = 1 ; row < 4 ; ++row )
		{
			int tmp = s_num_pts[row][NUM_WARPS_PER_BLOCK-1];
			if( lane_id < NUM_WARPS_PER_BLOCK )
				s_num_pts[row][lane_id] += sum;
			sum += tmp;
		}
	}
	__syncthreads();

}
int main()
{
	//parameters
	const int max_depth = 8;
	const int min_points_per_node = 20;
	int num_points = -1;

	//Set Cuda Device
  	int device_count = 0, device = -1, warp_size = 0;
  	checkCudaErrors( hipGetDeviceCount( &device_count ) );
	for( int i = 0 ; i < device_count ; ++i )
	{
		hipDeviceProp_t properties;
		checkCudaErrors( hipGetDeviceProperties( &properties, i ) );
		if( properties.major > 3 || ( properties.major == 3 && properties.minor >= 5 ) )
		{
		  device = i;
		  warp_size = properties.warpSize;
		  std::cout << "Running on GPU: " << i << " (" << properties.name << ")" << std::endl;
		  std::cout << "Warp Size: " << warp_size << std::endl;
		  std::cout << "Threads Per Block: " << properties.maxThreadsPerBlock<< std::endl;
		  break;
		}
		std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
	}
	if( device == -1 )
	{
		//cdpQuadTree requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...
		exit(EXIT_SUCCESS);
	}
	hipSetDevice(device);
	//Read Points from file and put it into x0(X points) and y0(Y Points)
	std::list<float> stlX, stlY;
	std::ifstream source("2.5width_4patels.txt");
	if(source.is_open()){
		int i = 0;
		for(std::string line;std::getline(source, line); i+=1)   //read stream line by line
		{
			std::istringstream in(line);      
			float x, y;
			in >> x >> y;       
			stlX.push_back(x);
			stlY.push_back(y);
		}
	}
	else{
		printf("No");
		exit(1);
	}

	num_points = stlX.size();	
	thrust::device_vector<float> x0( stlX.begin(), stlX.end() ); 
	thrust::device_vector<float> y0( stlY.begin(), stlY.end() );
	thrust::device_vector<float> x1( num_points );
	thrust::device_vector<float> y1( num_points );

	std::cout << num_points << std::endl;	
	
	//copy pointers to the points into the device because kernels don't support device_vector as input they accept raw_pointers
	//Thrust data types are not understood by a CUDA kernel and need to be converted back to its underlying pointer. 
	//host_points
	Points h_points[2];
	h_points[0].set( thrust::raw_pointer_cast( &x0[0] ), thrust::raw_pointer_cast( &y0[0] ) );
	h_points[1].set( thrust::raw_pointer_cast( &x1[0] ), thrust::raw_pointer_cast( &y1[0] ) );

	//device_points
	Points *d_points;
	checkCudaErrors( hipMalloc( (void**) &d_points, 2*sizeof(Points) ) ); 
	checkCudaErrors( hipMemcpy( d_points, h_points, 2*sizeof(Points), hipMemcpyHostToDevice ) );
	//Setting Cuda Heap size for dynamic memory allocation	
	size_t size = 1024*1024*1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, size);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);

	//Copy root node from host to device
	Quadtree_Node h_root;
	h_root.setRange(0, num_points);
	Quadtree_Node* d_root;
	checkCudaErrors( hipMalloc( (void**) &d_root, sizeof(Quadtree_Node)));
	checkCudaErrors( hipMemcpy( d_root, &h_root, sizeof(Quadtree_Node), hipMemcpyHostToDevice));

	//set the recursion limit based on max_depth
	//maximum possible depth is 24 levels
  	hipDeviceSetLimit( cudaLimitDevRuntimeSyncDepth, max_depth );

	getchar();
    return 0;
}

