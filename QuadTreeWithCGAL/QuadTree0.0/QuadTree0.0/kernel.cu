#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime_api.h>
#include <list> 
#include <vector> 
#include <sstream>
#include <fstream>
#include <string> 
#include <stdio.h>
#include <time.h>
#include <iostream> 


#include "global_datatype.h"
//#include "Clarkson-Delaunay.h"
#define FULL_MASK 0xffffffff
#define NUM_THREADS_PER_BLOCK 128 
#define STACK_SIZE 400
#define MAX_POINTS_SIZE 600 

class Points
{
	double *X;
	double *Y;
	int num_points;
public:
	__host__ __device__ Points() : X(NULL), Y(NULL), num_points(0) {}

	__host__ __device__ Points( double *x, double *y ) : X(x), Y(y) {}

	__host__ __device__ __forceinline__ double2 getPoint( int idx ) const
	{
		return make_double2( X[idx], Y[idx] );
	}
	__host__ __device__ __forceinline__ double2 getLastPoint() const
	{
		return make_double2( X[num_points-1], Y[num_points-1] );
	}
	__host__ __device__ __forceinline__ void printPoint( int idx ) const
	{
		printf("%f %f\n" ,X[idx] ,Y[idx]);
	}
	__host__ __device__ void copyPointsToHost(){
		double *newX = new double[MAX_POINTS_SIZE];
		double *newY = new double[MAX_POINTS_SIZE];
		checkCudaErrors( hipMemcpy( newX, X, MAX_POINTS_SIZE*sizeof(double), hipMemcpyDeviceToHost ) );
		checkCudaErrors( hipMemcpy( newY, Y, MAX_POINTS_SIZE*sizeof(double), hipMemcpyDeviceToHost ) );
		X = newX;
		Y = newY;
	}
	__host__ __device__ __forceinline__ void setPoint( int idx, const double2 &p ) 
	{
		X[idx] = p.x;
		Y[idx] = p.y;
	}
	__host__ __device__ void addPoint(double2 p){
		//For findInsidePoints we will use a Points class for each edge around which we
		//have to find the inside points.
		//Maximum possible result for findInsidePoints for an edge = MAX_POINTS_SIZE
		if(num_points == MAX_POINTS_SIZE){
			printf("Overflow\n\n");
			return;
		}		
		X[num_points] = p.x;
		Y[num_points] = p.y;
		num_points+=1;
	}
	__host__ __device__ int getNumberOfPoints(){
		return num_points;
	}
	__host__ __device__ __forceinline__ void set( double *x, double *y )
	{
		X = x;
		Y = y;
	}
};
class Indices{
	int* idx;
	int num_indices;
public:
	__host__ __device__ Indices() : idx(NULL), num_indices(0) {}
	__host__ __device__ void addIndex(int i){
		if(num_indices == MAX_POINTS_SIZE){
			printf("Overflow\n\n");
			return;
		}		
		// problem with idx for some reason
		idx[num_indices] = i;
		num_indices+=1;
	}
	__host__ __device__ int getNumberOfIndices(){
		return num_indices;
	}
	__host__ __device__ __forceinline__ int getIndex( int i) const
	{
		return idx[i];
	}
	__host__ __device__ void set(int* iptr){
		idx = iptr;
	}

};
class Adjacency_Matrix{
	int line_idx;
	int** adj_mat;
	int num_points;
	Points *points;
	double obtuse_angle; // default value 1.57 (pi / 2)
public: 
	__host__ __device__ Adjacency_Matrix() :line_idx(-1), num_points(0), adj_mat(NULL), points(NULL), obtuse_angle(1.57) {}
	__host__ __device__ Adjacency_Matrix(Points* pts) : obtuse_angle(1.57) {
		Points *points = pts;
		num_points = points->getNumberOfPoints();
		adj_mat = (int** )malloc(num_points*sizeof(int*));
		for(int i = 0;i<num_points;i++){
			adj_mat[i] = (int*)malloc(num_points*sizeof(int));
		}
		for(int i = 0;i<num_points;i++){
			for(int j = 0;j<num_points;j++){
				adj_mat[i][j] = 0;
			}
		}
	}
	//we are using Clarkson-Delaunay.h file for creating delaunay triangulations
	//scroll to end of that file to have a look at how the delaunay triangles are stored and how to print the triangles.
	//triangle_index_list stores the indices of the triangles 3 at a time
	__host__ __device__ Adjacency_Matrix( int lidx, unsigned int* triangle_index_list, int num_triangles, Points* pts): obtuse_angle(1.57){
		//here pts is the inside_points array from find inside points.
		//number of points there in the outer threshold for the particular line.
		line_idx = lidx;
		num_points = pts[lidx].getNumberOfPoints();
		points = &pts[lidx];
		//initialise adj_mat(an num_points X num_points matrix) with all 0
		adj_mat = (int** )malloc(num_points*sizeof(int*));
		for(int i = 0;i<num_points;i++){
			adj_mat[i] = (int*)malloc(num_points*sizeof(int));
		}
		for(int i = 0;i<num_points;i++){
			for(int j = 0;j<num_points;j++){
				adj_mat[i][j] = 0;
			}
		}

		for(int i = 0;i<num_triangles;i++){
			int a, b, c;
			a = triangle_index_list[i*3];
			b = triangle_index_list[i*3+1];
			c = triangle_index_list[i*3+2];
			addEdge(a,b);
			addEdge(a,c);
			addEdge(c,b);

		}
	}
	__host__ __device__ void addEdge(int i, int j){
		adj_mat[i][j] = 1;
		adj_mat[j][i] = 1;
	}
	__host__ __device__ double distance(int i, int j){
		double2 ipt = points->getPoint(i);
		double2 jpt = points->getPoint(j);
		double dist = (ipt.x - jpt.x)*(ipt.x - jpt.x) + (ipt.y - jpt.y)*(ipt.y - jpt.y);
		return dist;
	}
	//returns index of the closest point for a given point with index idx
	__host__ __device__ int getClosestPoint(int idx){
		int closest = -1;
		for(int j = 0;j<num_points;j++){
			if(j != idx && closest == -1)
				closest = j;
			else if(j != idx && distance(idx, closest) > distance(idx, j)){
				closest = j;
			}
		}	
		return closest;
	}
	//get closest point making an angle greater than obtuse_angle with idx and closest point
	__host__ __device__ int getClosestObtusePoint(int idx){
		int closest = getClosestPoint(idx);
		double2 P1 = points->getPoint(idx);
		double2 P2 = points->getPoint(closest);
		int obtuse_closest = -1;
		for(int i = 0;i<num_points;i++){
			if(i != idx && i != closest){
				double2 P3 = points->getPoint(i);
				double angle = atan2(P3.y - P1.y, P3.x - P1.x) - atan2(P2.y - P1.y, P2.x - P1.x);		
				if( angle > obtuse_angle || angle < -1*obtuse_angle){
					if(obtuse_closest == -1 ){
						obtuse_closest = i;
					}
					else if( distance(idx, obtuse_closest) > distance(idx, i)){
						obtuse_closest = i;
					}
				}
					
			}

		}
		return obtuse_closest;
	}
};

class Bounding_Box{
    double xMin , xMax, yMin, yMax;
public:
	__host__ __device__ Bounding_Box(){
		xMin = -700;
		yMin = -700;
		xMax = 700;
		yMax = 700;
	}
	__host__ __device__ double2 computeCenter(){
		double2 center;
		center.x = 0.5f * ( xMin + xMax);
		center.y = 0.5f * ( yMin + yMax );
		return center;
	}
	__host__ __device__ __forceinline__ double getxMax() const {
		return xMax;		
	}
	__host__ __device__ __forceinline__ double getyMax() const {
		return yMax;		
	}
	__host__ __device__ __forceinline__ double getyMin() const {
		return yMin;		
	}
	__host__ __device__ __forceinline__ double getxMin() const {
		return xMin;		
	}
	__host__ __device__ __forceinline__ void printBox() const {
		printf("%f %f %f %f ", xMin, yMax, xMax, yMax);
		printf("%f %f %f %f\n", xMax, yMin, xMin , yMin );
	}
	__host__ __device__ bool contains(const double2 &p) const {
		return (p.x >= xMin && p.y >= yMin && p.x < xMax && p.y < yMax);
	}
	__host__ __device__ void set(double x, double y, double X, double Y){
		xMin = x;
		yMin = y;
		xMax = X;
		yMax = Y;
	}
	__host__ __device__ int isInside(double x, double y){
		int res = (x <= xMax) && (x >= xMin) && (y <= yMax) && (y >= yMin);
		return res;
	}
};

class Quadtree_Node{
	//node index;
	int idx;
	Bounding_Box bb;
	//startIdx of points in the bb for the global data array
	int startIdx, endIdx;	
	Quadtree_Node *NE, *NW, *SW, *SE; 
public:
	__host__ __device__ Quadtree_Node() : idx(-1), startIdx(-1), endIdx(-1), NE(NULL), NW(NULL), SW(NULL), SE(NULL){

	}
	__host__ __device__ bool isNull(){
		return (idx == -1);
	}
	__host__ __device__ void setIdx(int idx){
		this->idx = idx;		
	}
	__host__ __device__ int getIdx(){
		return idx;
	}
	__host__ __device__ void setBoundingBox(double x,double y,double X,double Y){
		bb.set(x, y, X, Y);	
	}
	__host__ __device__ __forceinline__ Bounding_Box& getBoundingBox(){
		return bb;
	}
	__host__ __device__ void setRange(int s, int e){
		startIdx = s;
		endIdx = e;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getSW(){
		return SW;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getSE(){
		return SE;
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getNW(){
		return NW; 
	}
	__host__ __device__ __forceinline__ Quadtree_Node* getNE(){
		return NE; 
	}
	__host__ __device__ __forceinline__ void setSW( Quadtree_Node* ptr){
		SW = ptr;
	}
	__host__ __device__ __forceinline__ void setNW( Quadtree_Node* ptr){
		NW = ptr;
	}
	__host__ __device__ __forceinline__ void setSE( Quadtree_Node* ptr){
		SE = ptr;
	}
	__host__ __device__ __forceinline__ void setNE( Quadtree_Node* ptr){
		NE = ptr;
	}
	__host__ __device__ __forceinline__ int isLeaf(){
		return (NE == NULL);
	}
	__host__ __device__ __forceinline__ int getStartIdx(){
		return startIdx;
	}
	__host__ __device__ __forceinline__ int getEndIdx(){
		return endIdx;
	}
	__host__ __device__ __forceinline__ int numberOfPoints(){
		return endIdx - startIdx;
	}
};
//This class has to be mad error proof
//We have not yet handled when slope (M) is infinity
//make adjustments for this errors
class Line_Segment{
	double M, C;
	double2 P1, P2;
public:
	__host__ __device__ Line_Segment() : M(0.0), C(0.0), P1(make_double2(0.0, 0.0)), P2(make_double2(0.0, 0.0)){};
	__host__ __device__ Line_Segment(double2 p1, double2 p2){
		M = (p1.y - p2.y) / (p1.x - p2.x);
		C = p1.y - M*(p1.x);
		P1 = p1;
		P2 = p2;
	}
	__host__ __device__ Line_Segment(double m, double c){
		M = m;
		C = c;	
	}
	__host__ __device__ double getPerpendicularDistance(double2 p){
		double res = M*p.x - p.y + C;
		res = res / sqrt(1.0 + M*M);
		if(res < 0)
			res = (-1)*res;
		return res;
	}
	__host__ __device__ Line_Segment getLeftThreshold(double d){
		return Line_Segment(M, (C - d*sqrt(1.0 + M*M)));
	}
	__host__ __device__ Line_Segment getRightThreshold(double d){
		return Line_Segment(M, (C + d*sqrt(1.0 + M*M)));
	}
	__host__ __device__ int intersectsWithBox(Bounding_Box box){
		double minX = box.getxMin();
		double minX_Y = M*minX + C;
		double minY = box.getyMin();
		double minY_X = (minY - C ) / M;	
		double maxX = box.getxMax();
		double maxX_Y = M*maxX + C;
		double maxY = box.getyMax();
		double maxY_X = (maxY - C ) / M;	
		int res = box.isInside(minX, minX_Y) || box.isInside(minY_X, minY) ||box.isInside(maxX, maxX_Y) ||box.isInside(maxY_X, maxY);
		return res;
	}
	__host__ __device__ int insidePerpendicularBounds(Bounding_Box box){
		double perpM = -(1.00)*(1.0/M);
		double C1 = P1.y - perpM*P1.x;
		double C2 = P2.y - perpM*P2.x;
		double2 boundaries[4]; 
		boundaries[0] = make_double2(box.getxMin(), box.getyMax());
		boundaries[1] = make_double2(box.getxMax(), box.getyMax());
		boundaries[2] = make_double2(box.getxMin(), box.getyMin());
		boundaries[3] = make_double2(box.getxMax(), box.getyMin());
		for(int i = 0;i<4;i++){
			int sign1 = ((boundaries[i].y - perpM*boundaries[i].x - C1) >= 0);
			int sign2 = ((boundaries[i].y - perpM*boundaries[i].x - C2) >= 0);
			if(sign1 != sign2)
				return 1;
		}	
		return 0;
	}
};

class Parameters
{
public:
	const int min_points_per_node;
	//Introduced to minimise shifting of points
	//can have values only 0 and 1 based on slot
	//points[points_slot] is input slot
	//points[(points_slot+1)%2] is output slot
	int points_slot;
	__host__ __device__ Parameters( int mppn ) : min_points_per_node(mppn), points_slot(0) {}
	//copy constructor for the evaluation of children of current node
	__host__ __device__ Parameters( Parameters prm, bool ) : 
	min_points_per_node(prm.min_points_per_node), 
	points_slot((prm.points_slot+1)%2) 
	{}


};
class Quadtree_Stack
{
    private:
        Quadtree_Node* arr[STACK_SIZE];
        int top;
    public:
	__host__ __device__ Quadtree_Stack(){
		top=-1;
	}
	__host__ __device__ Quadtree_Node* push(Quadtree_Node* n){
	//check stack is full or not
		if(isFull()){
			return NULL;
		}
		++top;
		arr[top]=n;
		return n;
	}

	__host__ __device__ Quadtree_Node* pop(){
		//to store and print which number
		//is deleted
		Quadtree_Node* temp;
		//check for empty
		if(isEmpty())
			return NULL;
		temp=arr[top];
		--top;
		return temp;
		
	}
	__host__ __device__ int isEmpty(){
		if(top==-1)
			return 1;
		else
			return 0;   
	}

	__host__ __device__ int isFull(){
		if(top==(STACK_SIZE-1))
			return 1;
		else
			return 0;
	}
};
__device__ int nodeInsideThreshold(Line_Segment line, Bounding_Box box, Quadtree_Node* root,double threshold){
	double2 LT, RT, LB, RB;
	LT = make_double2(box.getxMin(), box.getyMax());
	RT = make_double2(box.getxMax(), box.getyMax());
	LB = make_double2(box.getxMin(), box.getyMin());
	RB = make_double2(box.getxMax(), box.getyMin());
	int boxInside = ((line.getPerpendicularDistance(LT) <= threshold) 
					|| (line.getPerpendicularDistance(RT) <= threshold)
					|| (line.getPerpendicularDistance(LB) <= threshold) ||
					(line.getPerpendicularDistance(RB) <= threshold));
	int boxIntersects = ((line.intersectsWithBox(box)) || (line.getLeftThreshold(threshold).intersectsWithBox(box)) || (line.getRightThreshold(threshold).intersectsWithBox(box)));
	return line.insidePerpendicularBounds(box) && (boxInside || boxIntersects);

}
__global__ void findOuterThresholdPoints(Quadtree_Node *root, Points *points,Line_Segment *lines, Points *inside_points, double threshold){
	int line_idx = threadIdx.x;
	Quadtree_Stack qst;
	qst.push(root);
	while(!qst.isEmpty()){
		Quadtree_Node* X = qst.pop();
		Bounding_Box box = X->getBoundingBox();
		if(X->isLeaf()){
			if(X->numberOfPoints() != 0){

				for(int i = X->getStartIdx(); i<X->getEndIdx(); i++){
					double2 p = points[0].getPoint(i);
						
					// printf("%f %f\n", line_idx, p.x, p.y);
					inside_points[line_idx].addPoint(p);

				}
			}
		}
		else{
			
		
			//nodeInsideThreshold returns 1 if the bounding box is inside the threshold or if the the box intersects any of the threshold lines or the line itself
			if( nodeInsideThreshold(lines[line_idx], box, X->getNE(), threshold) ){
				qst.push(X->getNE());
			}
			if(nodeInsideThreshold(lines[line_idx], box, X->getNW(), threshold)){
				qst.push(X->getNW());
			}
			if(nodeInsideThreshold(lines[line_idx], box, X->getSE(), threshold)){
				qst.push(X->getSE());
			}
			if(nodeInsideThreshold(lines[line_idx], box, X->getSW(), threshold)){
				qst.push(X->getSW());
			}
		}
	}
}
__global__ void findInnerThresholdIndices(Line_Segment *lines, Points *inside_points, Indices* inner_indices, double threshold){
	int line_idx = threadIdx.x;
	int num_points = inside_points[line_idx].getNumberOfPoints();
	for(int i = 0;i<num_points;i++){
		double2 p = inside_points[line_idx].getPoint(i);
		if(lines[line_idx].getPerpendicularDistance(p) <= threshold)	
		{
			inside_points[line_idx].printPoint(i);
			printf("%f %f\n", p.x, p.y);
			// printf("\n%d %d\n", line_idx, inner_indices[line_idx].getNumberOfIndices());
			inner_indices[line_idx].addIndex(i);
		}
	}
}
//the following things are done in the findIntersectionPoint function
//creating Delaunay triangulations of outer_threshold points for each line
//creating adjacency matrix for each delaunay triangulation
//unfortunately the delaunay triangulation function (BuildTriangleIndexList) used uses some pointers that are run on the CPU.
//WE NEED TO SEARCH FOR SOME OTHER DELAUNAY TRIANGULATION FUNCTION
//THE CONSTRUCTOR FOR ADJACENCY MATRIX IS BASED ON THE DELAUNAY TRIANGULATION WHICH WE HAD
//NEW CONSTRUCTOR will have to be written
__global__ void findIntersectionPoint(Line_Segment *lines, Points *inside_points, Indices* inner_indices,  int num_of_lines){
	int line_idx = threadIdx.x;
	int precision = 1000;
	//converting inside points to the required format for delaunay triangulation
	Points* pts = &inside_points[line_idx];
	int num_inside_points = pts->getNumberOfPoints();
	int* ptList = (int* )malloc(2*num_inside_points*sizeof(int));
	for(int i = 0, j = 0;i<num_inside_points;i++, j+=2){
		double2 p = pts->getPoint(i);
		ptList[j] = (int)p.x*precision;
		ptList[j+1] = (int)p.y*precision;
	}
	int num_triangles;
	unsigned int *triangle_index_list;
	// triangle_index_list = BuildTriangleIndexList(ptList, 0, num_inside_points, 2, 1, &num_triangles);
	Adjacency_Matrix adj(line_idx, triangle_index_list, num_triangles, pts); 

	//this stores the NNCrust Line segments
	Adjacency_Matrix curve(pts);

	//applying NNCrust and storing the new edges in the curve
	int num_indices = inner_indices[line_idx].getNumberOfIndices();
	for(int i = 0;i<num_indices;i++){
		int closest = curve.getClosestPoint(i);
		int obtuse_closest = curve.getClosestObtusePoint(i);
		curve.addEdge(i, closest);
		curve.addEdge(i, obtuse_closest);
		
	}
}
__global__ void buildQuadtree( Quadtree_Node *root, Points *points, Parameters prmtrs){
	const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warpSize;

	//shared memory
	extern __shared__ int smem[];
	
	//warp_id and lane_id
	const int warp_id = threadIdx.x / warpSize;
	const int lane_id = threadIdx.x % warpSize;
	
	// Addresses of shared Memory
	volatile int *s_num_pts[4];
	for( int i = 0 ; i < 4 ; ++i )
		s_num_pts[i] = (volatile int *) &smem[i*NUM_WARPS_PER_BLOCK];

	
	int NUM_POINTS = root->numberOfPoints();
	Bounding_Box &box = root->getBoundingBox();
	//stop recursion if num_points <= minimum number of points required for recursion 
	if( NUM_POINTS <= prmtrs.min_points_per_node){
		//If in current iteration the points are in slot 1
		//shift them to slot 0
		//we want the output in the slot 0
		if(prmtrs.points_slot == 1)
		{
			int it = root->getStartIdx(), end = root->getEndIdx();
			for( it += threadIdx.x; it < end ; it += NUM_THREADS_PER_BLOCK){
				if( it < end )
					points[0].setPoint(it, points[1].getPoint(it));
			}
		}
		
		return;
	}

	//get Center of the bounding box
	double2 center;
	center = box.computeCenter();
	//accomadate the excess points
	int NUM_POINTS_PER_WARP = max( warpSize, ( NUM_POINTS + NUM_WARPS_PER_BLOCK - 1 ) / NUM_WARPS_PER_BLOCK );
	
	int warp_begin = root->getStartIdx() + warp_id*NUM_POINTS_PER_WARP;
	int warp_end = min(warp_begin + NUM_POINTS_PER_WARP, root->getEndIdx());

	//reset counts of warps
	if( lane_id == 0 )
	{
		s_num_pts[0][warp_id] = 0;
		s_num_pts[1][warp_id] = 0;
		s_num_pts[2][warp_id] = 0;
		s_num_pts[3][warp_id] = 0;

	}
	
	//input points
	const Points &input = points[prmtrs.points_slot];
	
	//__any_sync(unsigned mask, predicate):
		//Evaluate predicate for all non-exited threads in mask and return non-zero if and only if predicate evaluates to non-zero for any of them.
	//count points in each warp that belong to which child
	for( int itr = warp_begin + lane_id ; __any(itr < warp_end ) ; itr += warpSize){
		bool is_active = itr < warp_end;
		//get the coordinates of the point
		double2 curP;
		if(is_active)
			curP = input.getPoint(itr);
		else
			curP = make_double2(0.0f, 0.0f);

		//consider standard anticlockwise quadrants for numbering 0 to 3

		//__ballot_sync(unsigned mask, predicate):
			//Evaluate predicate for all non-exited threads in mask and return an integer whose Nth bit is set if and only if predicate evaluates to non-zero for the Nth thread of the warp and the Nth thread is active.
		//__popc
			//Count the number of bits that are set to 1 in a 32 bit integer.
		//top-right Quadrant (Quadrant - I)
		bool pred = is_active && curP.x >= center.x && curP.y >= center.y;
		int curMask = __ballot( pred );
		int cnt = __popc( curMask );
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[0][warp_id] += cnt;

		//top-left Quadrant (Quadrant - II)
		pred = is_active && curP.x < center.x && curP.y >= center.y;
		curMask = __ballot(pred);
		cnt = __popc( curMask );
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[1][warp_id] += cnt;

		//bottom-left Quadrant (Quadrant - III)
		pred = is_active && curP.x < center.x && curP.y < center.y;
		curMask = __ballot(pred);
		cnt = __popc( curMask );
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[2][warp_id] += cnt;

		//bottom-right Quadrant (Quadrant - IV)
		pred = is_active && curP.x >= center.x && curP.y < center.y;
		curMask = __ballot(pred);
		cnt = __popc( curMask );
		if( cnt > 0 && lane_id == 0 )
			s_num_pts[3][warp_id] += cnt;
	}		

	//sychronize warps
	//__syncthreads() acts as a barrier at which all threads in the block must wait before any is allowed to proceed
	__syncthreads();
/*  	if(threadIdx.x == NUM_THREADS_PER_BLOCK - 1 && root->getIdx() == 1024){
		printf("Quadrant I : %d, %d \n", s_num_pts[0][0], s_num_pts[0][NUM_WARPS_PER_BLOCK-1]);
		for(int i = 0;i<NUM_WARPS_PER_BLOCK;i++){
			printf("%d ", s_num_pts[0][i]);
		}
		printf("\nQuadrant II : %d, %d \n", s_num_pts[1][0], s_num_pts[1][NUM_WARPS_PER_BLOCK-1]);
		for(int i = 0;i<NUM_WARPS_PER_BLOCK;i++){
			printf("%d ", s_num_pts[1][i]);
		}
		printf("\nQuadrant III : %d, %d \n", s_num_pts[2][0], s_num_pts[2][NUM_WARPS_PER_BLOCK-1]);
		for(int i = 0;i<NUM_WARPS_PER_BLOCK;i++){
			printf("%d ", s_num_pts[2][i]);
		}
		printf("\nQuadrant IV : %d, %d \n", s_num_pts[3][0], s_num_pts[3][NUM_WARPS_PER_BLOCK-1]);
		for(int i = 0;i<NUM_WARPS_PER_BLOCK;i++){
			printf("%d ", s_num_pts[3][i]);
		}
		printf("\n\n\n");
	} 
	__syncthreads(); */
 
	

	// Scan the warps' results to know the "global" numbers.
	// First 4 warps scan the numbers of points per child (inclusive scan).
	// In the later code we have used warp id to select the quadrant and lane_id to select a warp.
	if( warp_id < 4 )
	{
		int num_pts = lane_id < NUM_WARPS_PER_BLOCK ? s_num_pts[warp_id][lane_id] : 0;
		#pragma unroll
		for( int offset = 1 ; offset < NUM_WARPS_PER_BLOCK ; offset *= 2 )
		{

			//T __shfl_up_sync(unsigned mask, T var, unsigned int delta, int width=warpSize);	
			int n = __shfl_up(num_pts, offset, NUM_WARPS_PER_BLOCK );

			if( lane_id >= offset )
				num_pts += n;
		}
		if( lane_id < NUM_WARPS_PER_BLOCK )
			s_num_pts[warp_id][lane_id] = num_pts;
	}
	//after this we will have the local offsets, i.e , if we have a warp with id X
	//then s_num_pts[0][x] will store the number of points having warp id <= x 
	//and belong to the 0th quadrant
	__syncthreads();
	// Compute global offsets.
	//here lane_id will index the warps
	if( warp_id == 0 )
	{
		int sum = s_num_pts[0][NUM_WARPS_PER_BLOCK-1];
		for( int row = 1 ; row < 4 ; ++row )
		{
			int tmp = s_num_pts[row][NUM_WARPS_PER_BLOCK-1];
			if( lane_id < NUM_WARPS_PER_BLOCK )
				s_num_pts[row][lane_id] = s_num_pts[row][lane_id] + sum;
			sum += tmp;
		}
	}
	__syncthreads();
	//after this we have the global offsets, i.e, if warp id is X and quadrant q
	//then s_num_pts[q][x] will store the number of points having warp id <= x 
	//and belong to the quadrant <= q
	
	//make the Scan independent of the quadtree node you are currently in.
	// for this we just have to add the number of points that come before processing of the current node.
	if(threadIdx.x < 4*NUM_WARPS_PER_BLOCK){
		int val = (threadIdx.x == 0) ? 0 : smem[threadIdx.x - 1];
		smem[threadIdx.x] = val + root->getStartIdx();
	}
	__syncthreads();
	//move points to the next slot
	Points &output = points[(prmtrs.points_slot+1)%2];

	//Mask for threads in a warp that are less than the current lane_id
	int lane_mask_lt = (1 << lane_id) - 1; 
	// Move Points to the appropriate slot 
	// Quadtree sort implementation

	for( int itr = warp_begin + lane_id ; __any(itr < warp_end ) ; itr += warpSize){
		bool is_active = itr < warp_end;

		double2 curP;
		if(is_active){
			curP = input.getPoint(itr);
		}
		else{
			curP = make_double2(0.0f, 0.0f);
		}
		
		//counting QUADRANT I points
		bool pred =is_active && curP.x >= center.x && curP.y >= center.y;
		int curMask = __ballot(pred);
		int cnt = __popc( curMask & lane_mask_lt );
		int dest = s_num_pts[0][warp_id] + cnt;
		if( pred )
			output.setPoint(dest, curP);
		if( lane_id == 0 )
			s_num_pts[0][warp_id] += __popc( curMask);
		
		//countin QUADRANT II points
		pred = is_active && curP.x < center.x && curP.y >= center.y;
		curMask = __ballot(pred);
		cnt = __popc(curMask & lane_mask_lt);
		dest = s_num_pts[1][warp_id] + cnt;
		if(pred)
			output.setPoint(dest, curP);
		if( lane_id == 0)
			s_num_pts[1][warp_id] += __popc( curMask );

		//countin QUADRANT III points
		pred = is_active && curP.x < center.x && curP.y < center.y;
		curMask = __ballot(pred);
		cnt = __popc(curMask & lane_mask_lt);
		dest = s_num_pts[2][warp_id] + cnt;
		if(pred)
			output.setPoint(dest, curP);
		if( lane_id == 0)
			s_num_pts[2][warp_id] += __popc( curMask);

		//countin QUADRANT IV points
		pred = is_active && curP.x >= center.x && curP.y < center.y;
		curMask = __ballot(pred);
		cnt = __popc(curMask & lane_mask_lt);
		dest = s_num_pts[3][warp_id] + cnt;
		if(pred)
			output.setPoint(dest, curP);
		if( lane_id == 0)
			s_num_pts[3][warp_id] += __popc( curMask );

	}
	__syncthreads();
	//last thread will launch new block 
	if( threadIdx.x == NUM_THREADS_PER_BLOCK-1){
		//create children for next level
		// set index, bb, startIdx, endIdx and NE, NW, SE, SW children.
		//Index is used just for sake of future extension if some changes are required then
		//children nodes
		// std::cout << "( " << box.getxMin() << "," << box.getyMin() << ") , (" << box.getxMax() << "," << box.getyMax() << ") " << std::endl;	
		//print top left and top right points
		Quadtree_Node* NEC = (Quadtree_Node*)malloc(sizeof(Quadtree_Node));
		Quadtree_Node* NWC = (Quadtree_Node*)malloc(sizeof(Quadtree_Node));
		Quadtree_Node* SWC = (Quadtree_Node*)malloc(sizeof(Quadtree_Node));
		Quadtree_Node* SEC = (Quadtree_Node*)malloc(sizeof(Quadtree_Node));
		//set Bounding Box
		//printf("Center: %f %f\n", center.x, center.y);
		NEC->setBoundingBox(center.x, center.y, box.getxMax(), box.getyMax());
		NWC->setBoundingBox(box.getxMin(), center.y, center.x, box.getyMax());
		SWC->setBoundingBox(box.getxMin(), box.getyMin(), center.x, center.y);
		SEC->setBoundingBox(center.x, box.getyMin(), box.getxMax(), center.y);

		//set the start and end ranges
		//print the range of indices for children
/* 		printf("(%d, %d), ", root->getStartIdx(), s_num_pts[0][warp_id]);
		printf("(%d, %d), ", s_num_pts[0][warp_id], s_num_pts[1][warp_id]);
		printf("(%d, %d), ", s_num_pts[1][warp_id], s_num_pts[2][warp_id]);
		printf("(%d, %d)\n", s_num_pts[2][warp_id], s_num_pts[3][warp_id]);
 */

		NEC->setRange(root->getStartIdx(), s_num_pts[0][warp_id]);
		NWC->setRange(s_num_pts[0][warp_id], s_num_pts[1][warp_id]);
		SWC->setRange(s_num_pts[1][warp_id], s_num_pts[2][warp_id]);
		SEC->setRange(s_num_pts[2][warp_id], s_num_pts[3][warp_id]);

		//set the root children 
		root->setNE(NEC);
		root->setNW(NWC);
		root->setSW(SWC);
		root->setSE(SEC);

		//launch children
		buildQuadtree<<<1, NUM_THREADS_PER_BLOCK, 4*NUM_WARPS_PER_BLOCK*sizeof(int)>>>(NEC, points, Parameters(prmtrs, true));
		buildQuadtree<<<1, NUM_THREADS_PER_BLOCK, 4*NUM_WARPS_PER_BLOCK*sizeof(int)>>>(NWC, points, Parameters(prmtrs, true));
		buildQuadtree<<<1, NUM_THREADS_PER_BLOCK, 4*NUM_WARPS_PER_BLOCK*sizeof(int)>>>(SWC, points, Parameters(prmtrs, true));
		buildQuadtree<<<1, NUM_THREADS_PER_BLOCK, 4*NUM_WARPS_PER_BLOCK*sizeof(int)>>>(SEC, points, Parameters(prmtrs, true));
	}
}

__global__ void printQuadtree( Quadtree_Node *root){
	Bounding_Box box = root->getBoundingBox();
	box.printBox();

	if(root->getNE() != NULL){
		printQuadtree<<<1, 1>>>(root->getNE());
		printQuadtree<<<1, 1>>>(root->getNW());
		printQuadtree<<<1, 1>>>(root->getSE());
		printQuadtree<<<1, 1>>>(root->getSW());

	}

}
__global__ void printPoints( Points *inside_points, int num_of_lines){
	for(int i =0;i<num_of_lines;i++){
		int num_points= inside_points[i].getNumberOfPoints();
		for(int j = 0;j<num_points;j++){
			double2 p = inside_points[i].getPoint(j);
			printf("%f %f\n", p.x, p.y);
		}
	}
}

Points* initializeInsidePoints(int num_of_edges){
	Points *h_points = new Points[num_of_edges]; //= (Points*)malloc(num_of_edges*sizeof(Points));
	vector< thrust::device_vector<double> > X(num_of_edges);
	vector< thrust::device_vector<double> > Y(num_of_edges);
	

	for(int i = 0;i<num_of_edges;i++){
		X[i].resize(MAX_POINTS_SIZE);
		Y[i].resize(MAX_POINTS_SIZE);
		h_points[i].set(thrust::raw_pointer_cast(&X[i][0]), thrust::raw_pointer_cast(&Y[i][0])) ;
	}

	//device_points
	Points *d_points;
	checkCudaErrors( hipMalloc( (void**) &d_points, num_of_edges*sizeof(Points) ) ); 
	checkCudaErrors( hipMemcpy( d_points, h_points, num_of_edges*sizeof(Points), hipMemcpyHostToDevice ) );
	return d_points;	
}
Indices* initializeInnerIndices(int num_of_edges){

	Indices *h_inner_indices = new Indices[num_of_edges];
	vector< int* > idx( num_of_edges ); 

	for(int i = 0;i<num_of_edges;i++){
		idx[i] = (int*)malloc(MAX_POINTS_SIZE*sizeof(int));
		checkCudaErrors( hipMalloc( (void**) &idx[i], MAX_POINTS_SIZE*sizeof(int)) );
		h_inner_indices[i].set(idx[i]);
	}

	Indices *d_inner_indices;
	checkCudaErrors( hipMalloc( (void**) &d_inner_indices, num_of_edges*sizeof(Indices) ) ); 
	checkCudaErrors( hipMemcpy( d_inner_indices, h_inner_indices, num_of_edges*sizeof(Indices), hipMemcpyHostToDevice ) );

	return d_inner_indices;	
}

void create_Delaunay(Delaunay& dt, std::vector<Point_2> &input)
{
	dt.insert(input.begin(), input.end());
}
int main()
{

	std::string inputFile = "2.5width_4patels.txt";
	std::string outputFile = "InnerPoints(2.5width_4patels.txt).txt";
	freopen(outputFile.c_str() , "w", stdout);
	const int max_depth = 10;
	const int min_points_per_node = 5; // Min points per node
	int num_points = -1;

	//Read Points from file and put it into x0(X points) and y0(Y Points)
	std::vector<Point_2> OriginalSample, RandomSample;
	clock_t start = clock();
	std::list<double> stlX, stlY;
	std::ifstream source(inputFile);
	if(source.is_open()){
		int i = 0;
		for (std::string line; std::getline(source, line); i += 1)   //read stream line by line
		{
			std::istringstream in(line);
			double x, y;
			in >> x >> y;
			Point_2 original(x,y);
			OriginalSample.push_back(original);
			stlX.push_back(x);
			stlY.push_back(y);
		}
	}
	else{
		printf("No");
		exit(1);
	}
	/*
	std::ifstream input("neha1.txt");
	int num_of_points = 0;
	std::string data;
	while (getline(input, data))
	{
		Point_2 original;
		std::istringstream stream(data);
		while (stream >> original)
		{
			OriginalSample.push_back(original);
			++num_of_points;
		}
	}
	*/
	clock_t end = clock();
	double run_time = ((double)(end - start) / CLOCKS_PER_SEC);
	std::cout << "File Reading Time: " << run_time << std::endl;
	num_points = stlX.size();
	std::cout << "Number of Points: " << num_points << std::endl;

	//Delaunay Triangulations sample code using CGAL	
/* 	for(int i = 0; i<7; i++)
	{
		int n = std::rand() % (num_points - 1);
		
		RandomSample.push_back(OriginalSample.at(n));
		//if(outputRandomSample.is_open()){outputRandomSample<<OriginalSample.at(n)<<std::endl;}
	}



	//Creating Delaunay Triangulation of the points in the inputRandomSample
	Delaunay dt_sample;
	create_Delaunay(dt_sample, RandomSample);
	//void create_Voronoi(Delaunay& dt, std::vector<Ray_2>& ray, std::vector<Segment_2>& seg, std::vector<EdgeInfo>& ray_edges, std::vector<EdgeInfo>& seg_edges);

	//bool iterate=true;
	for (int i = 0; i < RandomSample.size(); i++){
		std::cout << RandomSample[i] << std::endl;
	}
 */
	//Set Cuda Device
	int device_count = 0, device = -1, warp_size = 0;
	checkCudaErrors(hipGetDeviceCount( &device_count ) );
	for( int i = 0 ; i < device_count ; ++i )
	{
		hipDeviceProp_t properties;
		checkCudaErrors( hipGetDeviceProperties( &properties, i ) );
		if( properties.major > 3 || ( properties.major == 3 && properties.minor >= 5 ) )
		{
			device = i;
			warp_size = properties.warpSize;
			// std::cout << "Running on GPU: " << i << " (" << properties.name << ")" << std::endl;
			// std::cout << "Warp Size: " << warp_size << std::endl;
			// std::cout << "Threads Per Block: " << properties.maxThreadsPerBlock<< std::endl;
			break;
		}
		std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
	}
	if( device == -1 )
	{
		//cdpQuadTree requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...
		exit(EXIT_SUCCESS);
	}
	hipSetDevice(device);
	
	start = clock();
	hipFree(0);
	end = clock();
	run_time = ((double)(end - start)/CLOCKS_PER_SEC);
	std::cout << "hipFree Time: " << run_time << std::endl;

	start = clock();
	thrust::device_vector<double> x0( stlX.begin(), stlX.end() ); 
	thrust::device_vector<double> y0( stlY.begin(), stlY.end() );
	thrust::device_vector<double> x1( num_points );
	thrust::device_vector<double> y1( num_points );
	end = clock();
	run_time = ((double)(end - start)/CLOCKS_PER_SEC);
	std::cout << "Data Conversion Time: " << run_time << std::endl;
	
	//copy pointers to the points into the device because kernels don't support device_vector as input they accept raw_pointers
	//Thrust data types are not understood by a CUDA kernel and need to be converted back to its underlying pointer. 
	//host_points(h for host, d for device)
	Points h_points[2];
	h_points[0].set( thrust::raw_pointer_cast( &x0[0] ), thrust::raw_pointer_cast( &y0[0] ) );
	h_points[1].set( thrust::raw_pointer_cast( &x1[0] ), thrust::raw_pointer_cast( &y1[0] ) );


	//device_points
	Points *d_points;
	checkCudaErrors( hipMalloc( (void**) &d_points, 2*sizeof(Points) ) ); 
	checkCudaErrors( hipMemcpy( d_points, h_points, 2*sizeof(Points), hipMemcpyHostToDevice ) );
	end = clock();
	run_time = ((double)(end - start)/CLOCKS_PER_SEC);
	std::cout << "GPU Data Transfer Time: " << run_time << std::endl; 
	
	//Setting Cuda Heap size for dynamic memory allocation	
	size_t size = 1024*1024*1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, size);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);

	//Copy root node from host to device
	Quadtree_Node h_root;
	h_root.setRange(0, num_points);
	h_root.setIdx(1024);
	Quadtree_Node* d_root;
	checkCudaErrors( hipMalloc( (void**) &d_root, sizeof(Quadtree_Node)));
	checkCudaErrors( hipMemcpy( d_root, &h_root, sizeof(Quadtree_Node), hipMemcpyHostToDevice));

	//set the recursion limit based on max_depth
	//maximum possible depth is 24 levels
	hipDeviceSetLimit( cudaLimitDevRuntimeSyncDepth, max_depth );
	Parameters prmtrs( min_points_per_node );
	const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warp_size;
	const int SHARED_MEM_SIZE = 4*NUM_WARPS_PER_BLOCK*sizeof(int);
	start = clock();
	buildQuadtree<<<1, NUM_THREADS_PER_BLOCK, SHARED_MEM_SIZE>>>(d_root, d_points, prmtrs);
	hipDeviceSynchronize();
	end = clock();
	run_time = ((double)(end - start)/CLOCKS_PER_SEC);
	std::cout << "Kernel Execution Time: " << run_time << std::endl; 
	checkCudaErrors( hipGetLastError() );
 	// printQuadtree<<<1,1>>>(d_root);
	int num_of_lines = 4;
	printf("Before Inside Initialization\n");
	Points* d_inside_points = initializeInsidePoints(num_of_lines);
	printf("After Inside points\n");
	Line_Segment *h_lines = new Line_Segment[num_of_lines];
	h_lines[0] = Line_Segment(make_double2(100.0, -200.0), make_double2(0.0, 300.0));
	h_lines[1] = Line_Segment(make_double2(0.0, 300.0), make_double2(600.0, 650.0));
	h_lines[2] = Line_Segment(make_double2(0.0, 300.0), make_double2(-550.0, 680.0));
	h_lines[3] = Line_Segment(make_double2(100.0, -200.0), make_double2(-600.0, -650.0));

	Line_Segment* d_lines;
	checkCudaErrors( hipMalloc( (void**) &d_lines, num_of_lines*sizeof(Line_Segment) ) ); 
	checkCudaErrors( hipMemcpy( d_lines, h_lines, num_of_lines*sizeof(Line_Segment), hipMemcpyHostToDevice ) );
	double threshold = 10.0;
	findOuterThresholdPoints<<<1,num_of_lines>>>(d_root, d_points, d_lines, d_inside_points, threshold);
	
	hipDeviceSynchronize();
//	printPoints<<<1,1>>>(d_inside_points, num_of_lines);
	Indices *d_inner_indices = initializeInnerIndices(num_of_lines);
		
	findInnerThresholdIndices <<<1, num_of_lines >>>(d_lines, d_inside_points, d_inner_indices, 20);
	hipDeviceSynchronize();
    return 0;
}