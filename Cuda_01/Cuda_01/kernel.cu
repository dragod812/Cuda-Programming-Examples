#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int a, int b, int *c){
	*c = a + b;
}

__global__ void addMatrix(int *c, int *a, int *b){
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	*(c + blockDim.y*i + j) = *(a + blockDim.y*i + j) + *(b + blockDim.y*i + j);
}
hipError_t addMatrixWC(int *C, int *A, int *B, int N);
/*
//addition of two numbers
int main()
{
int c;
int *dev_c;
hipError_t cudaStatus;
cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
if (cudaStatus != hipSuccess){
fprintf(stderr, "hipMalloc Failed!");
}
add << <1, 1 >> >(2, 7, dev_c);
cudaStatus = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess){
fprintf(stderr, "hipMemcpy Failed");
}
printf("ans %d", c);
getch();
hipFree(dev_c);
return 0;
}
*/
int main()
{   
	int N;
	scanf("%d", &N);
	int *A = (int *)malloc(N*N*sizeof(int));
	int *B = (int *)malloc(N*N*sizeof(int));
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			scanf("%d", (A + N*i + j));
		}
	}
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			scanf("%d", (B + N*i + j));
		}
	}


	int *C = (int *)malloc(N*N*sizeof(int));
	hipError_t cudaStatus = addMatrixWC(C, A, B, N);
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%d ", *(C + N*i + j));
		}
		printf("\n");
	}
	getch();
    return 0;
}
hipError_t addMatrixWC(int *C, int *A, int *B, int N){
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	
	hipError_t cudaStatus;
	printf("Inside Addmatrix!\n A:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%d ", *(A + N*i + j));
		}
		printf("\n");
	}
	printf("B:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%d ", *(B + N*i + j));
		}
		printf("\n");
	}
	cudaStatus = hipMalloc((void**)&dev_a, N*N*sizeof(int));
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, N*N*sizeof(int));
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_c, N*N*sizeof(int));
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, A, N*N*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_b, B, N*N*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	dim3 blockspergrid(N / 4, N / 4, 1);
	dim3 threadsperblock(4, 4, 1);
	addMatrix<<< blockspergrid, threadsperblock >> >(dev_c, dev_a, dev_b);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	
	cudaStatus = hipMemcpy(C, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
